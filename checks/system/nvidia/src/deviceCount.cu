
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <nvml.h>

#ifndef __USE_POSIX
   #define __USE_POSIX
#endif

#include <limits.h>

int main() {
   char hostname[HOST_NAME_MAX];
   unsigned int nvmlCount = 0;
   int deviceCount = 0;
   int result;
   nvmlReturn_t nvml_return;
   hipError_t error;

   result = gethostname(hostname, HOST_NAME_MAX);
   if (result != 0) {
       printf("error retriving the hostname\n");
       return -1;
   }

   nvml_return = nvmlInit_v2();
   if (nvml_return != NVML_SUCCESS) {
       printf("%s: error initializing NVML\n", hostname);
       return -1;
   }

   nvml_return = nvmlDeviceGetCount_v2(&nvmlCount);
   if (nvml_return != NVML_SUCCESS) {
       printf("%s: NVML error retrieving the device count:\n", hostname);
       nvml_return = nvmlShutdown();
       return -1;
   }

   error = hipGetDeviceCount(&deviceCount);
   if (error != hipSuccess) {
       printf("%s: CUDA error retrieving the device count:\n", hostname);
       nvml_return = nvmlShutdown();
       return -1;
   }

   if (deviceCount != nvmlCount) {
       printf("%s: NVML device count %d != CUDA device count: %d\n", hostname,
              nvmlCount, deviceCount);
       nvml_return = nvmlShutdown();
       return -1;
   }

   nvml_return = nvmlShutdown();
   printf("%s: NVML device count == Cuda device count == %d\n", hostname,
          deviceCount);

   return 0;
}
